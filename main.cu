#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include <glad/gl.h>
#include <GLFW/glfw3.h>
#include "MeshVao.h"
#include "Shader.h"
#include "Sphere.h"
#include <vector>
#include <iostream>

GLFWwindow *window;

#define spring_Y 3e4
#define dashpot_damping 1e4
#define drag_damping 1

unsigned int N = 128;
float quad_size = 1.0 / N;

float ball_radius = 0.3;
float3 ball_center = make_float3(0, 0, 0);
float dt = 0.04 / N;
int sub_step = (1.0f / 60.0f) / dt;

float3 *x, *v;

int2 *spring_offset;

__global__ void init_kernel(float3 *_x, float3 *_v,
                            int N, float quad_size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x > N || y > N)
        return;
    _x[x + N * y] = make_float3(x * quad_size - 0.5, 0.6, y * quad_size - 0.5);
    _v[x + N * y] = make_float3(0, 0, 0);
}

__global__ void init_offset(int2 *offset)
{
    int cnt = 0;
    for (int i = -1; i < 2; i++)
    {
        for (int j = -1; j < 2; j++)
        {
            if (!(i == 0 && j == 0))
            {
                offset[cnt] = make_int2(i, j);
                cnt++;
            }
        }
    }
    offset[cnt++] = make_int2(0, 2);
    offset[cnt++] = make_int2(2, 0);
    offset[cnt++] = make_int2(0, -2);
    offset[cnt++] = make_int2(-2, 0);
}

__global__ void gravity_kernel(float3 *_v, float dt, int N)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x > N || y > N)
        return;
    _v[x + y * N] += make_float3(0, -9.8, 0) * dt;
}

__global__ void update_position_kernel(float3 *_x, float3 *_v, float dt, int N)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x > N || y > N)
        return;

    _x[x + y * N] += _v[x + y * N] * dt;
}

__global__ void spring_kernel(float3 *_x, float3 *_v, int2 *offset,
                              float quad_size, float dt, int N)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x > N || y > N)
        return;
    for (int i = 0; i < 12; i++)
    {
        float3 force = make_float3(0, 0, 0);
        int other_x = x + offset[i].x;
        int other_y = y + offset[i].y;
        if (other_x >= 0 && other_x < N && other_y >= 0 && other_y < N)
        {
            float3 x_ij = _x[x + N * y] - _x[other_x + N * other_y];
            float3 v_ij = _v[x + N * y] - _v[other_x + N * other_y];

            float3 d = normalize(x_ij);
            float current_length = norm3df(x_ij.x, x_ij.y, x_ij.z);
            float original_length = quad_size * norm3df(offset[i].x, offset[i].y, 0);
            force += -spring_Y * d * (current_length / original_length - 1);
            force += -dot(v_ij, d) * d * dashpot_damping * quad_size;
        }
        _v[x + N * y] += force * dt;
    }
    _v[x + N * y] *= exp2f(drag_damping * dt);
}

__global__ void collision_kernel(float3 *_x, float3 *_v,
                                 float3 ball_center, float ball_radius,
                                 float dt, int N)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x > N || y > N)
        return;

    float3 delta_x = _x[x + N * y] - ball_center;
    float sdf = norm3df(delta_x.x, delta_x.y, delta_x.z) - ball_radius;
    if (sdf <= 0)
    {
        float3 d = normalize(delta_x);
        _v[x + N * y] -= __min(dot(d, _v[x + N * y]), 0.0f) * d;
    }
}

void step()
{
    gravity_kernel<<<{(N + 7) / 8, (N + 7) / 8}, {8, 8}>>>(v, dt, N);
    spring_kernel<<<{(N + 7) / 8, (N + 7) / 8}, {8, 8}>>>(x, v, spring_offset, quad_size, dt, N);

    collision_kernel<<<{(N + 7) / 8, (N + 7) / 8}, {8, 8}>>>(x, v, ball_center, ball_radius, dt, N);

    update_position_kernel<<<{(N + 7) / 8, (N + 7) / 8}, {8, 8}>>>(x, v, dt, N);
}

__host__ void init_glfw()
{
    glfwInit();

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    window = glfwCreateWindow(1920, 1080, "ClothSim", NULL, NULL);
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, [](GLFWwindow *window, int width, int height)
                                   { glViewport(0, 0, width, height); });
    gladLoadGL(glfwGetProcAddress);
}

int main()
{
    init_glfw();

    Shader shader("./shader/shader.vs", "./shader/shader.fs");
    std::vector<float3> ball_pos;
    std::vector<unsigned int> ball_index;
    make_sphere(ball_pos,ball_index,ball_center,ball_radius);

    MeshVao ball_vao(ball_pos.size(),ball_index.size());
    ball_vao.copyIn(ball_pos.data(),nullptr,nullptr,ball_index.data(),
    ball_pos.size()*sizeof(float3),0,0,ball_index.size()*sizeof(unsigned int));

    hipMalloc(&x, N * N * sizeof(float3));
    hipMalloc(&v, N * N * sizeof(float3));
    hipMalloc(&spring_offset, 12 * sizeof(int2));

    init_kernel<<<{(N + 7) / 8, (N + 7) / 8}, {8, 8}>>>(x, v,
                                                        N, quad_size);
    init_offset<<<1, 1>>>(spring_offset);

    std::vector<unsigned int> index;

    for (unsigned int i = 0; i < N - 1; i++)
    {
        for (unsigned int j = 0; j < N - 1; j++)
        {
            index.push_back(i + j * N);
            index.push_back(i + 1 + j * N);
            index.push_back(i + (j + 1) * N);
            index.push_back(i + 1 + j * N);
            index.push_back(i + (j + 1) * N);
            index.push_back(i + 1 + (j + 1) * N);
        }
    }

    MeshVao vao(N * N, index.size());

    float3 *cpu_x = (float3 *)malloc(N * N * sizeof(float3));

    int cnt = 0;
    while (true)
    {
        if (glfwWindowShouldClose(window))
            exit(0);
        glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        cnt++;
        if (cnt == 128)
        {
            cnt %= 16;
            init_kernel<<<{(N + 7) / 8, (N + 7) / 8}, {8, 8}>>>(x, v,
                                                                N, quad_size);
        }
        for (int i = 0; i < sub_step; i++)
        {
            step();
        }
        hipMemcpy(cpu_x, x, N * N * sizeof(float3), hipMemcpyDeviceToHost);

        vao.copyIn(cpu_x, nullptr, nullptr, index.data(),
                   N * N * sizeof(float3), 0, 0, index.size() * sizeof(unsigned int));
        shader.use();
        shader.setMat4("Project", glm::perspective(glm::radians(45.0f), 1920.0f / 1080.0f, 0.1f, 100.0f));
        shader.setMat4("View", glm::lookAt(glm::vec3(0, 0, 3), glm::vec3(0, 0, 0), glm::vec3(0, 1, 0)));
        vao.draw();
        ball_vao.draw();
        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    free(cpu_x);

    hipFree(x);
    hipFree(v);
    return 0;
}